#include "hip/hip_runtime.h"
#include "binMethod.h"
#include "utilMethods.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>     /* srand, rand */
#include <math.h>

#define DP_SIZE  12288
__device__ double costFuncGPU(int R1, int R2){
	return R1*R2;
}
__device__ int gpuBitCount(unsigned int n){
	unsigned int count = 0;
	while (n)
	{
		count += n & 1;
		n >>= 1;
	}
	return count;
}
__global__ void subSetKernel(int *table, int *sets, int size){

	if (threadIdx.x == 0){
		__syncthreads();
	}
	else if(threadIdx.x < size +1){
		int setS, subS1, subS2, cost2;
		int bitS, bitS2;
		bool first_run;
		//Using THREAD INDEX as Array ENtry
		setS = sets[threadIdx.x-1];

		bitS = gpuBitCount(setS);
		//calculating the subset S1
		subS1 = setS & (-setS);

		//while the child subset is different from the intiale calcualte the mirror subset
		first_run = true;
		while (subS1 != setS){
			subS2 = setS - subS1;
			bitS2 = gpuBitCount(subS2);
			//for left/right deeop tree the subset bit size has to bit 1 less from the set
			if (bitS2 == (bitS - 1)){
				//for intiiale run intiate the set values
				int c1 = table[subS1 * 3];
				int c2 = table[subS2 * 3];
				if (first_run){
					table[setS * 3] = costFuncGPU(c1, c2);
					  table[setS * 3 + 1] = subS1;
					  table[setS * 3 + 2] = subS2;
					  first_run = false;
				}else{
					//once better path found set this one to the dp table
					cost2 = costFuncGPU(c1, c2);
					if (table[setS] > cost2){
						table[setS * 3] = cost2;
						table[setS * 3 + 1] = subS1;
						table[setS * 3 + 2] = subS2;
					}
				}
			}
			subS1 = setS & (subS1 - setS);
		}

	}
	__syncthreads();
}

hipError_t runOnGpu();

int main(){

	int input = 0x0FFF;
	int input_count = countSetBits(input);

	//input tables
	int* sql_input = new int[input_count];
	for (int i = 0; i <= input_count; i++){
		sql_input[i] = rand() % 300 + 1;
	}
	//double sql_sel[5] = { 0.01, 0.34, 0.55, 0.28, 0.88 };
	int bitNumber = countSetBits(input);
	int dp_table_size = 3 * (int)pow(2.0, bitNumber );
	int* dp_table = new int[dp_table_size];

	// [SIZE ][CHILD_1 ][CHILD_2];

	hipError_t cudaStatus;
	// Allocate GPU buffer
	int *dev_sel;
	int *dev_table;
	cudaStatus = hipMalloc((void**)&dev_sel, 5 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_table, dp_table_size*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Levels enumeration
	int mask = 0x01;
	int currBitSet, coeff, bitSet, table;
	for (int j = 1; j <= bitNumber; j++) {
		bitSet = mask;
		currBitSet = countSetBits(bitSet);
		//setting up the first level = importing the initale tables caridnality
		if (currBitSet == 1){
			table = getTableIndex(currBitSet);
			dp_table[currBitSet * 3] = sql_input[table];
			for (int i = 0; i < bitNumber -1; i++){
				currBitSet = next_set_of_n_elements(currBitSet);
				table = getTableIndex(currBitSet);
				dp_table[currBitSet * 3] = sql_input[table];
			}
		}
		else{
			//getting the coefficient
			if (currBitSet != bitNumber){
				coeff = binomialeCoeff(currBitSet, bitNumber);
			}
			else{
				coeff = 1;
			};

			//getting array of all sets to treat;
			int *all_sets = new int[coeff*sizeof(int)];
			all_sets[0] = bitSet;
			for (int i = 0; i < coeff - 1; i++){
				bitSet = next_set_of_n_elements(bitSet);
				all_sets[i + 1] = bitSet;
			};
			printf("Binom Coefficient is: %d ->", coeff);
			if (coeff > 400){
				printf(" GPU run \n");
				//enumerating subsets on CPU/GPU -> occupancy 
				int *dev_sets;
					//Allocating needed memory 
				cudaStatus = hipMalloc((void**)&dev_sets, coeff*sizeof(int));
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "hipMalloc failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(dev_sets, all_sets, coeff*sizeof(int), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "cudaCopy_in_sets failed!");
					goto Error;
				}
				cudaStatus = hipMemcpy(dev_table, dp_table, dp_table_size*sizeof(int), hipMemcpyHostToDevice);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "cudaCopy_in_table failed!");
					goto Error;
				}
					//Launching the kernel
				subSetKernel <<<1, coeff+1>>>(dev_table, dev_sets, coeff);
					//Copying back to host
				cudaStatus = hipMemcpy(dp_table, dev_table, dp_table_size*sizeof(int), hipMemcpyDeviceToHost);
				if (cudaStatus != hipSuccess) {
					fprintf(stderr, "cudaCopy_ou_table failed!");
					goto Error;
				}
					//Freeing cuda allocatade mem
				hipFree(dev_sets);
			}
			else {
				printf(" CPU run\n");
					//running the subset enumaration on CPU
				getSubSets(dp_table, all_sets, coeff);
			}
			delete[] all_sets;
			all_sets = NULL;
		}
		mask = mask << 1;
		mask = mask + 1;
	}
	hipFree(dev_table);
	hipFree(dev_sel);

	printResult(dp_table, input);

Error:
	hipFree(dev_table);
	hipFree(dev_table);
	hipFree(dev_sel);
	return cudaStatus;
}

hipError_t runOnGpu();